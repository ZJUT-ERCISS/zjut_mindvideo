#include <string.h>
#include <torch/extension.h> // 头文件引用部分
#include "ms_ext.h"
#include "deform_conv.h"
#include <stdio.h>

extern "C" int msDCN_backward_input(
    int nparam,
    void** params,
    int* ndims,
    int64_t** shapes,
    const char** dtypes,
    void* stream,
    void* extra
){
    // printf("start bprop input in cuda\n");
    hipStream_t custream = static_cast<hipStream_t>(stream);
    hipStreamSynchronize(custream);
    std::vector<int> vecp;
    vecp.push_back(0);
    vecp.push_back(1);
    vecp.push_back(2);
    vecp.push_back(3);
    vecp.push_back(4);
    vecp.push_back(5);
    vecp.push_back(6);
    // printf("tensor index get\n");
    
    auto tensors = get_torch_tensors(nparam, params, ndims, shapes, dtypes, vecp, c10::kCUDA);

    // printf("tensor list get\n");

    auto input = tensors[0];
    auto offset = tensors[1];
    auto grad_output = tensors[2];
    auto grad_input = tensors[3];
    auto grad_offset = tensors[4];
    auto weight = tensors[5];
    auto columns = tensors[6];

    // printf("backward_input tensor get\n");
    int *device_kw = (int *)(params[7]);
    int *device_kh = (int *)(params[8]);
    int *device_dw = (int *)(params[9]);
    int *device_dh = (int *)(params[10]);
    int *device_padw = (int *)(params[11]);
    int *device_padh = (int *)(params[12]);
    int *device_dilationw = (int *)(params[13]);
    int *device_dilationh = (int *)(params[14]);
    int *device_group = (int *)(params[15]);
    int *device_deformable_group = (int *)(params[16]);
    int *device_im2col_step = (int *)(params[17]);

    int *host_kw = new int[1]();
    int *host_kh = new int[1]();
    int *host_dw = new int[1]();
    int *host_dh = new int[1]();
    int *host_padw = new int[1]();
    int *host_padh = new int[1]();
    int *host_dilationw = new int[1]();
    int *host_dilationh = new int[1]();
    int *host_group = new int[1]();
    int *host_deformable_group = new int[1]();
    int *host_im2col_step = new int[1]();

    hipMemcpy(host_kw, device_kw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_kh, device_kh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dw, device_dw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dh, device_dh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_padw, device_padw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_padh, device_padh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dilationw, device_dilationw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dilationh, device_dilationh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_group, device_group, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_deformable_group, device_deformable_group, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_im2col_step, device_im2col_step, 1, hipMemcpyDeviceToHost);

    int kw = *host_kw;
    int kh = *host_kh;
    int dw = *host_dw;
    int dh = *host_dh;
    int padw = *host_padw;
    int padh = *host_padh;
    int dilationw = *host_dilationw;
    int dilationh = *host_dilationh;
    int group = *host_group;
    int deformable_group = *host_deformable_group;
    int im2col_step = *host_im2col_step;

    // printf("backward_input param get\n");

    deform_conv_backward_input(input,
                               offset,
                               grad_output,
                               grad_input,
                               grad_offset,
                               weight,
                               columns,
                               kw,
                               kh,
                               dw,
                               dh,
                               padw,
                               padh,
                               dilationw,
                               dilationh,
                               group,
                               deformable_group,
                               im2col_step);
    
    // printf("backward_input done\n");
    // float *output1 = (float *)(params[3]);    
    // float *output2 = (float *)(params[18]);
    // size_t size1 = sizeof(float);
    // for(int i = 0; i < ndims[3]; i++){
    //     size1 *= shapes[3][i];
    //     // printf("%ld %ld\n", shapes[3][i], size);
    // }

    // float *output3 = (float *)(params[4]);    
    // float *output4 = (float *)(params[19]);
    // size_t size2 = sizeof(float);
    // for(int i = 0; i < ndims[4]; i++){
    //     size2 *= shapes[4][i];
    //     // printf("%ld %ld\n", shapes[3][i], size);
    // }



    // hipMemcpy(output2, output1, size1, hipMemcpyDeviceToDevice);
    // hipMemcpy(output4, output3, size2, hipMemcpyDeviceToDevice);
    delete[] host_kw;
    delete[] host_kh;
    delete[] host_dw;
    delete[] host_dh;
    delete[] host_padw;
    delete[] host_padh;
    delete[] host_dilationw;
    delete[] host_dilationh;
    delete[] host_group;
    delete[] host_deformable_group;
    delete[] host_im2col_step;

    return 0;

}

extern "C" int msDCN_backward_filter(
    int nparam,
    void** params,
    int* ndims,
    int64_t** shapes,
    const char** dtypes,
    void* stream,
    void* extra
){
    // printf("start bprop filter in cuda\n");
    hipStream_t custream = static_cast<hipStream_t>(stream);
    hipStreamSynchronize(custream);
    std::vector<int> vecp;
    vecp.push_back(0);
    vecp.push_back(1);
    vecp.push_back(2);
    vecp.push_back(3);
    vecp.push_back(4);
    vecp.push_back(5);

    auto tensors = get_torch_tensors(nparam, params, ndims, shapes, dtypes, vecp, c10::kCUDA);

    auto input = tensors[0];
    auto offset = tensors[1];
    auto gradOutput = tensors[2];
    auto gradWeight = tensors[3];
    auto columns = tensors[4];
    auto ones = tensors[5];

    // printf("start bprop filter tensor get\n");

    int *device_kw = (int *)(params[6]);
    int *device_kh = (int *)(params[7]);
    int *device_dw = (int *)(params[8]);
    int *device_dh = (int *)(params[9]);
    int *device_padw = (int *)(params[10]);
    int *device_padh = (int *)(params[11]);
    int *device_dilationw = (int *)(params[12]);
    int *device_dilationh = (int *)(params[13]);
    int *device_group = (int *)(params[14]);
    int *device_deformable_group = (int *)(params[15]);
    float *device_scale = (float *)(params[16]);
    int *device_im2col_step = (int *)(params[17]);

    int *host_kw = new int[1]();
    int *host_kh = new int[1]();
    int *host_dw = new int[1]();
    int *host_dh = new int[1]();
    int *host_padw = new int[1]();
    int *host_padh = new int[1]();
    int *host_dilationw = new int[1]();
    int *host_dilationh = new int[1]();
    int *host_group = new int[1]();
    int *host_deformable_group = new int[1]();
    float *host_scale = new float[1]();
    int *host_im2col_step = new int[1]();

    hipMemcpy(host_kw, device_kw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_kh, device_kh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dw, device_dw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dh, device_dh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_padw, device_padw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_padh, device_padh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dilationw, device_dilationw, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_dilationh, device_dilationh, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_group, device_group, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_deformable_group, device_deformable_group, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_scale, device_scale, 1, hipMemcpyDeviceToHost);
    hipMemcpy(host_im2col_step, device_im2col_step, 1, hipMemcpyDeviceToHost);

    int kw = *host_kw;
    int kh = *host_kh;
    int dw = *host_dw;
    int dh = *host_dh;
    int padw = *host_padw;
    int padh = *host_padh;
    int dilationw = *host_dilationw;
    int dilationh = *host_dilationh;
    int group = *host_group;
    int deformable_group = *host_deformable_group;
    float scale = *host_scale;
    int im2col_step = *host_im2col_step;

    // printf("start bprop filter param get\n");

    deform_conv_backward_filter(input,
                                offset,
                                gradOutput,
                                gradWeight,
                                columns,
                                ones,
                                kw,
                                kh,
                                dw,
                                dh,
                                padw,
                                padh,
                                dilationw,
                                dilationh,
                                group,
                                deformable_group,
                                scale,
                                im2col_step);
        
    // float *output1 = (float *)(params[2]); 
    // float *output2 = (float *)(params[18]);
    // size_t size = sizeof(float);
    // for(int i = 0; i < ndims[3]; i++){
    //     size *= shapes[3][i];
    //     // printf("%ld %ld\n", shapes[3][i], size);
    // }

    // hipMemcpy(output2, output1, size, hipMemcpyDeviceToDevice);
    // printf("start bprop filter done\n");
    delete[] host_kw;
    delete[] host_kh;
    delete[] host_dw;
    delete[] host_dh;
    delete[] host_padw;
    delete[] host_padh;
    delete[] host_dilationw;
    delete[] host_dilationh;
    delete[] host_group;
    delete[] host_deformable_group;
    delete[] host_scale;
    delete[] host_im2col_step;

    return 0;

}